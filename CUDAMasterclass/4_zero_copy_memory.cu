#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "common.h"

// the below two kernels are identical
// memory bound application
__global__ void sumArrays(int *A, int *B, int *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

__global__ void sumArraysZeroCopy(int *A, int *B, int *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

// sudo /usr/local/cuda/bin/nv-nsight-cu-cli --section MemoryWorkloadAnalysis zero_copy_mem
int main(int argc, char **argv)
{
	// set up device
	int dev = 0;
	hipSetDevice(dev);

	// get device properties
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	// check if support mapped memory
	if (!deviceProp.canMapHostMemory)
	{
		printf("Device %d does not support mapping CPU host memory!\n", dev);
		hipDeviceReset();
		exit(EXIT_SUCCESS);
	}

	// set up data size of vectors
	int power = 22;

	if (argc > 1) power = atoi(argv[1]);

	int nElem = 1 << power;
	size_t nBytes = nElem * sizeof(int);

	// part 1: using device memory
	// malloc host memory
	int *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (int *)malloc(nBytes);
	h_B = (int *)malloc(nBytes);
	hostRef = (int *)malloc(nBytes); // seems not used
	gpuRef = (int *)malloc(nBytes); // copy summed result to host

	// initialize data at host side
	initialize(h_A, nElem,INIT_ONE_TO_TEN);
	initialize(h_B, nElem);
	memset(gpuRef, 0, nBytes);

	// malloc device global memory
	int *d_A, *d_B, *d_C;
	hipMalloc((int**)&d_A, nBytes);
	hipMalloc((int**)&d_B, nBytes);
	hipMalloc((int**)&d_C, nBytes);

	// transfer data from host to device
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	// set up execution configuration
	// each thread block has 512 threads, 16 warps
	// each grid in total has 8192 thread blocks
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);

	sumArrays << <grid, block >> >(d_A, d_B, d_C, nElem);
	
	hipDeviceSynchronize();
	// copy kernel result back to host side
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// free device global memory
	hipFree(d_A);
	hipFree(d_B);

	// free host memory
	free(h_A);
	free(h_B);

	// part 2: using zerocopy memory for array A and B
	// allocate zerocpy memory
	hipHostAlloc((void **)&h_A, nBytes, hipHostMallocMapped); // zero copy memory
	hipHostAlloc((void **)&h_B, nBytes, hipHostMallocMapped);

	// initialize data at host side
	initialize(h_A, nElem, INIT_ONE_TO_TEN);
	initialize(h_B, nElem, INIT_ONE_TO_TEN);
	memset(gpuRef, 0, nBytes); // set initial value

	// get the mapped device pointer
	// (void **)&d_A: first get address for d_A, then convert into type void **
	// essentially, the first arg needs type void **, so use &d_A to convert to this type
	hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0); // get pointer to zero copy memory
	hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);

	// execute kernel with zero copy memory
	sumArraysZeroCopy << <grid, block >> >(d_A, d_B, d_C, nElem);

	hipDeviceSynchronize();

	// copy kernel result back to host side
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// free  memory
	hipFree(d_C);
	hipHostFree(h_A); // need special free function
	hipHostFree(h_B);

	free(hostRef); // not used
	free(gpuRef);

	// reset device
	hipDeviceReset();
	return EXIT_SUCCESS;
}
