#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void stream_test_modified(int* in, int * out, int size)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid < size)
	{
		//THIS FOR LOOP IS ONLY FOR VISUALIZING PURPOSE  
		for (int i = 0; i < 25; i++)
		{
			out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
		}
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 18;
	int byte_size = size * sizeof(int);

	//initialize host pointer
	// add a second copy to see execution order for different streams
	int* h_in, *h_ref, *h_in2, *h_ref2;
	
	// first allocate pinned memory
	// seems to forget free pinned meory
	hipHostMalloc((void**)&h_in,byte_size);
	hipHostMalloc((void**)&h_ref, byte_size);
	hipHostMalloc((void**)&h_in2, byte_size);
	hipHostMalloc((void**)&h_ref2, byte_size);

	initialize(h_in, INIT_RANDOM);
	initialize(h_in2, INIT_RANDOM);


	//allocate device pointers
	int * d_in, *d_out, *d_in2, *d_out2;
	hipMalloc((void**)&d_in, byte_size);
	hipMalloc((void**)&d_out, byte_size);
	hipMalloc((void**)&d_in2, byte_size);
	hipMalloc((void**)&d_out2, byte_size);

	// create cuda streams
	hipStream_t str,str2;
	hipStreamCreate(&str);
	hipStreamCreate(&str2);


	//kernel launch
	dim3 block(128);
	dim3 grid(size / block.x);
	
	//transfer data from host to device
	// add to add stream as parameter
	hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice,str);
	// third parameter is for dynamic shared memory, no use, so set to 0
	stream_test_modified << <grid, block,0,str >> > (d_in, d_out, size);
	hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost,str);
	
	hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
	stream_test_modified << <grid, block, 0, str2 >> > (d_in2, d_out2, size);
	hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);
	
	// need to sync stream
	// essentially, use hipDeviceSynchronize should also be fine
	hipStreamSynchronize(str);
	// then destroy stream in the end
	hipStreamDestroy(str);

	hipStreamSynchronize(str2);
	hipStreamDestroy(str2);

	hipDeviceReset();
	return 0;
}