#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define c0 1
#define c1 2
#define c2 3
#define c3 4
#define c4 5

#define RADIUS 4

// block size
#define BDIM 128

//constant memory declaration, with __constant__ identifier
__constant__ int coef[9];

//stencil calculation in host side
// cpu computation for comparison
void host_const_calculation(int * in, int * out, int size)
{
	for (int i = 0; i < size; i++)
	{

		// handle boundary situation
		if (i < RADIUS)
		{
			out[i] = in[i + 4] * c0
				+ in[i + 3] * c1
				+ in[i + 2] * c2
				+ in[i + 1] * c3
				+ in[i] * c4;

			if (i == 3)
			{
				out[i] += in[2] * c3;
				out[i] += in[1] * c2;
				out[i] += in[0] * c1;
			}
			else if (i == 2)
			{
				out[i] += in[1] * c3;
				out[i] += in[0] * c2;
			}
			else if (i == 1)
			{
				out[i] += in[0] * c3;
			}
		}
		else if ((i + RADIUS) >= size)
		{
			out[i] = in[i - 4] * c0
				+ in[i - 3] * c1
				+ in[i - 2] * c2
				+ in[i - 1] * c3
				+ in[i] * c4;
			
			if (i == size - 4)
			{
				out[i] += in[size - 3] * c3;
				out[i] += in[size - 2] * c2;
				out[i] += in[size - 1] * c1;
			}
			else if (i == size -3)
			{
				out[i] += in[size - 2] * c3;
				out[i] += in[size - 1] * c2;
			}
			else if (i == size - 2)
			{
				out[i] += in[size - 1] * c3;
			}
		}
		else
		{
			// simple convolution operation
			out[i] = (in[i - 4] + in[i + 4])*c0
				+ (in[i - 3] + in[i + 3])*c1
				+ (in[i - 2] + in[i + 2])*c2
				+ (in[i - 1] + in[i + 1])*c3
				+ in[i] * c4;
		}
	}
}

//setting up constant memory from host
// it's essentially the convolution kernel
void setup_coef_1()
{
	const int h_coef[] = { c0,c1,c2,c3,c4,c3,c2,c1,c0 };
	// define variable through constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(coef), h_coef, (9) * sizeof(float));
}

__global__ void constant_stencil_smem_test(int * in, int * out, int size)
{
	//shared mem declaration
	// make sense, need 2-side padding
	// radius = ( stencil length -1 ) / 2
	// only used by certain thread block
	__shared__ int smem[BDIM + 2 * RADIUS]; // 128 + 4*2 = 136, 0~136

	// 0 ~ (1<<22-1)
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	// 1<<22/128
	int bid = blockIdx.x;
	// since 1d grid correspondings to 1d array
	// therefore, only the first and last thread block will need to do padding
	int num_of_blocks = gridDim.x;

	int value = 0;

	// global id
	if (gid < size)
	{
		//index with offset
		// caused by padding
		int sidx = threadIdx.x + RADIUS; // 4~132

		//load data to shared mem
		// using shared memory can speedup processing
		// MIDDLE OF ELEMENT
		// for each thread block, 4~132 indices store needed data from global memory
		smem[sidx] = in[gid];

		// following part is more like a data preprocessing step
		// normal case
		if (bid != 0 && bid != (num_of_blocks - 1))
		{
			// only need data within radius range, 0~4
			if (threadIdx.x < RADIUS)
			{
				// why only providing two sides' value are enough?
				// for each thread block, 0~4 indices store needed data from global memory as well
				smem[sidx - RADIUS] = in[gid - RADIUS];
				// sidx is 4~8, sidx + BDIM is 132~136
				// for each thread block, 132~136 indices store needed data from global memory as well
				smem[sidx + BDIM] = in[gid + BDIM];
			}
		}
		// handle boundary situation
		// left-side boundary
		else if (bid == 0)
		{
			if (threadIdx.x < RADIUS)
			{
				smem[sidx - RADIUS] = 0; // padding on left side
				smem[sidx + BDIM] = in[gid + BDIM];
			}
		}
		// right-side boundary
		else
		{
			if (threadIdx.x < RADIUS)
			{
				smem[sidx - RADIUS] = in[gid - RADIUS];
				smem[sidx + BDIM] = 0; // padding on right side
			}
		}

		// wait untill all the threads in block finish storing smem
		// after syncing, then all needed data are available in shared memory
		__syncthreads();

		// convolution operation
		// directly use variable in constant memory
		value += smem[sidx - 4] * coef[0];
		value += smem[sidx - 3] * coef[1];
		value += smem[sidx - 2] * coef[2];
		value += smem[sidx - 1] * coef[3];
		value += smem[sidx - 0] * coef[4];
		value += smem[sidx + 1] * coef[5];
		value += smem[sidx + 2] * coef[6];
		value += smem[sidx + 3] * coef[7];
		value += smem[sidx + 4] * coef[8];

		// each thread is responsible for each element's stencil computation result in a 1d array
		out[gid] = value;
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 22;
	int byte_size = sizeof(int) * size;
	int block_size = BDIM;

	int * h_in, *h_out, *h_ref;

	h_in = (int*)malloc(byte_size);
	h_out = (int*)malloc(byte_size);
	h_ref = (int*)malloc(byte_size);

	initialize(h_in, size, INIT_ONE);

	int * d_in, *d_out;
	hipMalloc((void**)&d_in, byte_size);
	hipMalloc((void**)&d_out, byte_size);

	hipMemcpy(d_in, h_in, byte_size, hipMemcpyHostToDevice);
	hipMemset(d_out, 0, byte_size);

	setup_coef_1();

	dim3 blocks(block_size);
	dim3 grid(size / blocks.x);

	constant_stencil_smem_test << < grid, blocks >> > (d_in, d_out, size);
	hipDeviceSynchronize();

	hipMemcpy(h_ref, d_out, byte_size, hipMemcpyDeviceToHost);

	host_const_calculation(h_in, h_out, size);

	compare_arrays(h_ref, h_out, size);

	hipFree(d_out);
	hipFree(d_in);
	free(h_ref);
	free(h_out);
	free(h_in);

	return 0;
}