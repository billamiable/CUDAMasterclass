#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define BDIMX 128 //64
#define BDIMY 8
#define IPAD 2

// this one is the same as read row write column in global memory section
__global__ void transpose_read_row_write_column_benchmark(int * mat, 
	int* transpose, int nx, int ny)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		//read by row, write by col
		transpose[ix * ny + iy] = mat[iy * nx + ix];
	}
}

__global__ void transpose_smem(int * in, int* out, int nx, int ny)
{
	// 2d shared memory
	// 64*8=512
	__shared__ int tile[BDIMY][BDIMX];

	//input index
	int ix, iy, in_index;

	//output index
	int i_row, i_col, _1d_index, out_ix, out_iy, out_index;

	//ix and iy calculation for input index
	ix = blockDim.x * blockIdx.x + threadIdx.x;
	iy = blockDim.y * blockIdx.y + threadIdx.y;

	//input index
	// agree
	in_index = iy * nx + ix;

	//1D index calculation fro shared memory
	_1d_index = threadIdx.y * blockDim.x + threadIdx.x;

	//col major row and col index calcuation
	// agree
	i_row = _1d_index / blockDim.y;
	i_col = _1d_index % blockDim.y;

	//coordinate for transpose matrix
	// agree
	out_ix = blockIdx.y * blockDim.y + i_col;
	out_iy = blockIdx.x * blockDim.x + i_row;

	//output array access in row major format
	// agree
	out_index = out_iy * ny + out_ix;

	if (ix < nx && iy < ny)
	{
		//load from in array in row major and store to shared memory in row major
		// step1: load data to shared memory
		tile[threadIdx.y][threadIdx.x] = in[in_index];

		//wait untill all the threads load the values
		// VIP
		__syncthreads();

		// step2: store data from shared memory
		// load from shared memory in column major and store to global memory in row major
		out[out_index] = tile[i_col][i_row];
	}
}

__global__ void transpose_smem_pad(int * in, int* out, int nx, int ny)
{
	// only difference is add IPAD here
	__shared__ int tile[BDIMY][BDIMX + IPAD];

	//input index
	int ix, iy, in_index;

	//output index
	int i_row, i_col, _1d_index, out_ix, out_iy, out_index;

	//ix and iy calculation for input index
	ix = blockDim.x * blockIdx.x + threadIdx.x;
	iy = blockDim.y * blockIdx.y + threadIdx.y;

	//input index
	in_index = iy * nx + ix;

	//1D index calculation fro shared memory
	_1d_index = threadIdx.y * blockDim.x + threadIdx.x;

	//col major row and col index calcuation
	i_row = _1d_index / blockDim.y;
	i_col = _1d_index % blockDim.y;

	//coordinate for transpose matrix
	out_ix = blockIdx.y * blockDim.y + i_col;
	out_iy = blockIdx.x * blockDim.x + i_row;

	//output array access in row major format
	out_index = out_iy * ny + out_ix;

	if (ix < nx && iy < ny)
	{
		//load from in array in row major and store to shared memory in row major
		tile[threadIdx.y][threadIdx.x] = in[in_index];

		//wait untill all the threads load the values
		__syncthreads();

		// load from shared memory in column major and store to global memory in row major
		out[out_index] = tile[i_col][i_row];
	}
}

__global__ void transpose_smem_pad_unrolling(int * in, int* out, int nx, int ny)
{
	// here use 1d array instead of 2d array
	// theorectically, it should be fine to directly use 2d array
	// unrolling factor is 2
	// IPAD only need to apply padding after unrolling
	__shared__ int tile[BDIMY * (2 * BDIMX + IPAD)];

	//input index
	int ix, iy, in_index;

	//output index
	int i_row, i_col, _1d_index, out_ix, out_iy, out_index;

	//ix and iy calculation for input index
	ix = 2 * blockDim.x * blockIdx.x + threadIdx.x; // ix is changed because unrolling is performed in row-major way
	iy = blockDim.y * blockIdx.y + threadIdx.y;

	//input index
	in_index = iy * nx + ix;

	//1D index calculation fro shared memory
	_1d_index = threadIdx.y * blockDim.x + threadIdx.x;

	//col major row and col index calcuation
	// if square matrix, i_row = threadIdx.y; i_col = threadIdx.x
	i_row = _1d_index / blockDim.y;
	i_col = _1d_index % blockDim.y;

	//coordinate for transpose matrix
	out_ix = blockIdx.y * blockDim.y + i_col;
	out_iy = 2 * blockIdx.x * blockDim.x + i_row; // same as before, but after transpose, need to change iy here

	//output array access in row major format
	out_index = out_iy * ny + out_ix;

	if (ix < nx && iy < ny)
	{
		// previously, using threadIdx.y and threadIdx.x is enough
		// also already changed from 2d array to 1d array
		// directly apply the new dim is fine
		int row_idx = threadIdx.y * (2 * blockDim.x + IPAD) + threadIdx.x;

		//load from in array in row major and store to shared memory in row major
		// unroll twice
		tile[row_idx] = in[in_index];
		tile[row_idx+ BDIMX] = in[in_index + BDIMX];  // get index for next thread block in x dim

		//wait untill all the threads load the values
		__syncthreads();

		// same as before
		// this calculation is different, because it's actually not using the same data operated above
		// but instead, due to the existence of syncthreads, code can operate any data
		// here it's accessing shared memory using column major way
		int col_idx = i_col * (2 * blockDim.x + IPAD) + i_row;

		// load from shared memory in column major and store to global memory in row major
		// unroll twice
		// essentially, i think this kind of operation can be done through triton
		out[out_index] = tile[col_idx];
		// make sense to have ny*BDIMX
		// as long as the operation can cover all the data, then it's fine
		out[out_index + ny* BDIMX] = tile[col_idx + BDIMX]; // small offset in input lead to large offset in output
	}
}

int main(int argc, char** argv)
{
	//default values for variabless
	int nx = 1024;
	int ny = 1024;
	int block_x = BDIMX;
	int block_y = BDIMY;
	int kernel_num = 0;

	//set the variable based on arguments
	if (argc > 1)
		nx = 1 << atoi(argv[1]);
	if (argc > 2)
		ny = 1 << atoi(argv[2]);
	if (argc > 3)
		block_x = 1 << atoi(argv[3]);
	if (argc > 4)
		block_y = 1 <<atoi(argv[4]);

	int size = nx * ny;
	int byte_size = sizeof(int*) * size;

	printf("Matrix transpose for %d X % d matrix with block size %d X %d \n",nx,ny,block_x,block_y);

	int * h_mat_array = (int*)malloc(byte_size);
	int * h_trans_array = (int*)malloc(byte_size);
	int * h_ref = (int*)malloc(byte_size);

	initialize(h_mat_array,size ,INIT_ONE_TO_TEN);

	//matirx transpose in CPU
	mat_transpose_cpu(h_mat_array, h_trans_array, nx, ny);

	int * d_mat_array, *d_trans_array;
	
	gpuErrchk(hipMalloc((void**)&d_mat_array, byte_size));
	gpuErrchk(hipMalloc((void**)&d_trans_array, byte_size));

	gpuErrchk(hipMemcpy(d_mat_array, h_mat_array, byte_size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(d_trans_array, 0, byte_size));

	dim3 blocks(block_x, block_y);
	dim3 grid(nx/block_x, ny/block_y);

	printf("Launching smem kernel \n");
	transpose_smem <<< grid, blocks>> > (d_mat_array,d_trans_array,nx, ny);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost));
	compare_arrays(h_ref, h_trans_array,size);

	printf("Launching benchmark kernel \n");
	hipMemset(d_trans_array,0, byte_size);
	transpose_read_row_write_column_benchmark << < grid, blocks >> > (d_mat_array, d_trans_array, nx, ny);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost));
	compare_arrays(h_ref, h_trans_array, size);

	printf("Launching smem padding kernel \n");
	hipMemset(d_trans_array, 0, byte_size);
	transpose_smem_pad << < grid, blocks >> > (d_mat_array, d_trans_array, nx, ny);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost));
	compare_arrays(h_ref, h_trans_array, size);

	printf("Launching smem padding and unrolling kernel \n");
	hipMemset(d_trans_array, 0, byte_size);

	// VIP: after unrolling, make sure the x dim of grid size is divided by 2
	grid.x = grid.x / 2;
	
	transpose_smem_pad_unrolling << < grid, blocks >> > (d_mat_array, d_trans_array, nx, ny);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost));
	compare_arrays(h_ref, h_trans_array, size);

	hipFree(d_trans_array);
	hipFree(d_mat_array);
	free(h_ref);
	free(h_trans_array);
	free(h_mat_array);

	gpuErrchk(hipDeviceReset());
	return EXIT_SUCCESS;
}