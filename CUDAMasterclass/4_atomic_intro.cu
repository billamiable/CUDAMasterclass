#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void incr(int *ptr)
{
	// due to no use of threadid here, so all threads are accessing the same memory address using broadcast
	// therefore below code has output as 1
	// if having multiple thread blocks, the output will be undefined (tested still 1)
	// int temp = *ptr;
	// temp = temp + 1;
	// *ptr = temp;

	// to enable processing between competing threads, use atomic operation
	// below code has output as 32
	atomicAdd(ptr,1);
}

int main()
{
	int value = 0;	
	int SIZE = sizeof(int);
	int ref = -1;

	int *d_val;
	hipMalloc((void**)&d_val, SIZE);
	hipMemcpy(d_val, &value, SIZE, hipMemcpyHostToDevice);
	incr << <1, 32 >> > (d_val);
	hipDeviceSynchronize();
	hipMemcpy(&ref,d_val,SIZE, hipMemcpyDeviceToHost);

	printf("Updated value : %d \n",ref);

	hipDeviceReset();
	return 0;
}