#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_unrolling_blocks2(int * input, int * temp, int size)
{
	int tid = threadIdx.x;

	// essentially, should be used to find the next data for calculation
	// because nearby thread block will only need 1 thread block to calculate
	// so, for each thread block that needs to calculate, it needs to access nearby thread block
	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;

	// global data index
	int index = BLOCK_OFFSET + tid;

	// same for the data that participated in calculation as above line
	int * i_data = input + BLOCK_OFFSET;

	// this is essentially thread block unrolling, not loop unrolling
	if ((index + blockDim.x) < size)
	{
		// first sum data from two consecutive thread blocks
		input[index] += input[index + blockDim.x];
	}

	// make sure the above part is done
	__syncthreads();

	// next part is the same as reduction interleaved pairs method
	for (int offset = blockDim.x / 2; offset > 0;offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}

		__syncthreads();
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}

__global__ void reduction_unrolling_blocks4(int * input, int * temp, int size)
{
	int tid = threadIdx.x;

	// only difference is 2->4
	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;

	int index = BLOCK_OFFSET + tid;

	int * i_data = input + BLOCK_OFFSET;

	//  similarily, here needs to do sum over 4 elements
	if ((index + 3 * blockDim.x) < size)
	{
		int a1 = input[index];
		int a2 = input[index + blockDim.x];
		int a3 = input[index+ 2* blockDim.x];
		int a4 = input[index+ 3 *blockDim.x];
		input[index] = a1 + a2 + a3 + a4;
	}

	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}

		__syncthreads();
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}


int main(int argc, char ** argv)
{
	printf("Running parallel reduction with unrolling blocks8 kernel \n");

	int size = 1 << 27;
	int byte_size = size * sizeof(int);
	int block_size = 128;

	int * h_input, *h_ref;
	h_input = (int*)malloc(byte_size);

	initialize(h_input, size, INIT_RANDOM);

	int cpu_result = reduction_cpu(h_input, size);

	dim3 block(block_size);
	// this part is important, for 2 unrolling, divide by 2, for 4 unrolling, divide by 4
	// dim3 grid((size / block_size) / 4);
	dim3 grid((size / block_size) / 2);

	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

	int temp_array_byte_size = sizeof(int)* grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int * d_input, *d_temp;
	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
		hipMemcpyHostToDevice));

	reduction_unrolling_blocks2 << < grid, block >> > (d_input, d_temp, size);
	// reduction_unrolling_blocks4 << < grid, block >> > (d_input, d_temp, size);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}

	compare_results(gpu_result, cpu_result);

	gpuErrchk(hipFree(d_input));
	gpuErrchk(hipFree(d_temp));
	free(h_input);
	free(h_ref);

	gpuErrchk(hipDeviceReset());
	return 0;
}