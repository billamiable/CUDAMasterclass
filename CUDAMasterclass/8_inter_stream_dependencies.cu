#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void k1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		printf("This is a test 1 \n");
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 15;

	hipStream_t stm1,stm2,stm3;
	hipStreamCreate(&stm1);
	hipStreamCreate(&stm2);
	hipStreamCreate(&stm3);

	hipEvent_t event1;
	// don't need to record timing, thus to improve performance
	hipEventCreateWithFlags(&event1, hipEventDisableTiming);

	dim3 block(128);
	dim3 grid(size / block.x);
	
	k1 << <grid, block, 0, stm1 >> > ();
	// create event after launching first kernel
	hipEventRecord(event1, stm1);
	// use wait operation to ask stream3 to wait event to finish
	// therefore, it's expected that stream3 kernel launch after stream1 kernel finishes
	hipStreamWaitEvent(stm3, event1, 0);

	k1 << <grid, block, 0, stm2 >> > ();
	k1 << <grid, block, 0, stm3 >> > ();

	// destroy event
	hipEventDestroy(event1);

	hipStreamDestroy(stm1);
	hipStreamDestroy(stm2);
	hipStreamDestroy(stm3);

	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
