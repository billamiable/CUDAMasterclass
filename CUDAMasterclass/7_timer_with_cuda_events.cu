#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void event_test()
{
	double sum = 0.0;
	for (int i = 0; i < 1000; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 12;

	dim3 block(128);
	dim3 grid(size / block.x);

	hipEvent_t start, end;

	// create events
	hipEventCreate(&start);
	hipEventCreate(&end);

	// push event to queue before kernel launch
	hipEventRecord(start);

	event_test << < grid,block >>> ();

	// push event to queue after kernel launch
	hipEventRecord(end);
	// sync to make sure event is finished
	// mainly to block host operation until event is reached and finished
	hipEventSynchronize(end);

	float time;
	// check time cost
	hipEventElapsedTime(&time, start, end);

	printf("Kernel execution time using events : %f \n",time);

	// destroy events
	hipEventDestroy(start);
	hipEventDestroy(end);

	hipDeviceReset();
	return 0;
}