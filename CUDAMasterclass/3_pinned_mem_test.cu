#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

int main(int argc, char **argv) 
{   								
	// memory size   128 MBs
	int isize = 1<<25;   
	int nbytes = isize * sizeof(float);
											
	// allocate the host memory   
    // option1: use pageable memory
	//float *h_a = (float *)malloc(nbytes);
    // option2: use pinned memory
	float *h_a;
	hipHostMalloc((float **)&h_a, nbytes, hipHostMallocDefault);

	// allocate the device memory   
	float *d_a; 
   hipMalloc((float **)&d_a, nbytes);
									
	// initialize the host memory   
	for(int i=0;i<isize;i++) 
		h_a[i] = 7;
									
	// transfer data from the host to the device   
	hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
									
	// transfer data from the device to the host   
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);
							
	// free memory   
	hipFree(d_a);
    // option1: use free to free pageable memory
	//free(h_a);
    // option2: use cudaFreeHost to free pinned memory
	hipHostFree(h_a);
									
	// reset device    
	hipDeviceReset();   
	return EXIT_SUCCESS;
}