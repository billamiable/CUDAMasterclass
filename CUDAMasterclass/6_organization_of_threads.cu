#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_threadIds()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d \n",
		threadIdx.x,threadIdx.y,threadIdx.z);
}

int main()
{
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8,8); // each block have 8 threads in x and y dim
	dim3 grid(nx/ block.x, ny/block.y); // 2*2 grid

	print_threadIds << <grid,block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
