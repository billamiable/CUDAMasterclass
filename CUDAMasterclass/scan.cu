#include "hip/hip_runtime.h"
#include "scan.cuh"

#include "common.h"

#define BLOCK_SIZE 512

// cpu implementation
void inclusive_scan_cpu(int *input, int *output, int size)
{
	output[0] = input[0];

	// make sense, just do prefix sum including index's element
	for (int i = 1; i < size; i++)
	{
		output[i] = output[i - 1] + input[i];
	}
}

void exclusive_scan_cpu(int *input, int *output, int size)
{
	output[0] = 0;

	// make sense, just do prefix sum including index's element
	for (int i = 1; i < size; i++)
	{
		output[i] = output[i - 1] + input[i-1];
	}
}

// naive idea implementation
__global__ void naive_inclusive_scan_single_block(int *input, int size)
{
	int tid = threadIdx.x;
	// only support 1 thread block, so gid should be identical to tid
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		// only first several elements contain final result
		// since tid == gid, so gid-stride should equal or larger than 0
		for (int stride = 1; stride <= tid; stride *= 2)
		{
			input[gid] += input[gid - stride];
			// make sure to sync in each iteration
			__syncthreads();
		}
	}
}

__global__ void efficient_inclusive_scan_single_block(int *input,int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		for (int stride = 1; stride <= tid; stride *= 2)
		{
			input[gid] += input[gid - stride];
			__syncthreads();
		}
	}
}

__global__ void efficient_exclusive_scan_single_block(int *input,int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		// upper-sweep phase (reduction)
		for (int stride = 1; stride < blockDim.x; stride *= 2)
		{
			// this time, do sum for odd indices
			// tid+1 is to make sure -1 won't cause less than 0 index
			// multiple by 2 is to make sure 2n-1
			int index = ( tid + 1 ) * 2 * stride - 1;
			if (index < blockDim.x) {
				input[index] += input[index - stride];
			}
			__syncthreads();
		}

		//set root value to 0
		if (tid == 0)
			input[blockDim.x - 1] = 0;
		
		int temp = 0;

		// down-sweep phase
		for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
		{
			// same as above
			int index = ( tid + 1 ) * 2 * stride - 1;
			if (index < blockDim.x) {
				temp = input[index - stride]; // assign left child to temp
				input[index - stride] = input[index]; // assign left child with parent node's value
				input[index] += temp; // sum left child's original value with parent node's value and assign to right child
			}
			__syncthreads();
		} 
	}
}


__global__ void efficient_inclusive_scan_single_block(int *input,int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		// upper-sweep phase (reduction)
		for (int stride = 1; stride < blockDim.x; stride *= 2)
		{
			// this time, do sum for odd indices
			// tid+1 is to make sure -1 won't cause less than 0 index
			// multiple by 2 is to make sure 2n-1
			int index = ( tid + 1 ) * 2 * stride - 1;
			if (index < blockDim.x) {
				input[index] += input[index - stride];
			}
			__syncthreads();
		}

		//set root value to 0
		if (tid == 0)
			input[blockDim.x - 1] = 0;
		
		int temp = 0;

		// down-sweep phase
		for (int stride = blockDim.x / 4; stride > 0; stride /= 2)
		{
			// same as above
			int index = ( tid + 1 ) * 2 * stride - 1;
			if (index + stride < blockDim.x) {
				// use intermediate result to calculcate prefix sum
				input[index + stride] += input[index];	
			}
			__syncthreads();
		} 
	}
}


__global__ void sum_aux_values(int *input,  int *aux, int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		for (int i = 0; i < blockIdx.x; i++)
		{
			input[gid] += aux[i];
			__syncthreads();
		}
	}
}

int main(int argc, char**argv)
{
	printf("Scan algorithm execution starterd \n");

	int input_size = 1 << 10; // 1024
	
	if (argc > 1)
	{
		input_size = 1 << atoi(argv[1]);
	}
	
	const int byte_size = sizeof(int) * input_size;

	int * h_input, *h_output, *h_ref, *h_aux;

	clock_t cpu_start, cpu_end, gpu_start, gpu_end;

	h_input = (int*)malloc(byte_size);
	h_output = (int*)malloc(byte_size);
	h_ref = (int*)malloc(byte_size);

	initialize(h_input, input_size, INIT_ONE);

	cpu_start = clock();
	// inclusive_scan_cpu(h_input, h_output, input_size);
	exclusive_scan_cpu(h_input, h_output, input_size);
	cpu_end = clock();

	int *d_input, *d_aux;
	hipMalloc((void**)&d_input, byte_size);

	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	dim3 block(BLOCK_SIZE); // 512
	dim3 grid(input_size/ block.x); // by default is 2

	int aux_byte_size = block.x * sizeof(int);
	hipMalloc((void**)&d_aux , aux_byte_size);

	h_aux = (int*)malloc(aux_byte_size);
	
	// this is incorrect because should only use 1 thread block in naive implementation
	// naive_inclusive_scan_single_block << <grid, block >> > (d_input, input_size);
	// naive_inclusive_scan_single_block << <1, 1024 >> > (d_input, input_size);
	// efficient_exclusive_scan_single_block << <1, 1024 >> > (d_input, input_size);
	efficient_inclusive_scan_single_block << <1, 1024 >> > (d_input, input_size);
	hipDeviceSynchronize();

	hipMemcpy(h_aux, d_aux, aux_byte_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_ref, d_input, byte_size, hipMemcpyDeviceToHost);

	print_arrays_toafile(h_ref, input_size, "input_array.txt");

	for (int i = 0; i < input_size; i++)
	{
		for (int j = 0; j < i / BLOCK_SIZE ; j++)
		{
			h_ref[i] += h_aux[j];
		}
	}

	print_arrays_toafile(h_aux,grid.x, "aux_array.txt");

	//sum_aux_values << < grid, block >> > (d_input, d_aux, input_size);
	//hipDeviceSynchronize();

	//hipMemcpy(h_ref, d_input, byte_size, hipMemcpyDeviceToHost );
	//print_arrays_toafile_side_by_side(h_ref, h_output, input_size, "scan_outputs.txt");

	compare_arrays(h_ref, h_output, input_size);

	gpuErrchk(hipDeviceReset());
	return 0;
}