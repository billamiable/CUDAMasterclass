#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 1024
// #define BLOCK_SIZE 128

// reduction with complete unrolling version
// use template to make more generalizable
template<unsigned int iblock_size>
__global__ void reduction_gmem_benchmark(int * input,int * temp, int size)
{
	int tid = threadIdx.x;
	int * i_data = input + blockDim.x * blockIdx.x;

	//manual unrolling depending on block size
	if (iblock_size >= 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];

	__syncthreads();

	if (iblock_size >= 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];

	__syncthreads();

	if (iblock_size >= 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];

	__syncthreads();

	if (iblock_size >= 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];

	__syncthreads();

	//unrolling warp
	if (tid < 32)
	{
		volatile int * vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}

template<unsigned int iblock_size>
__global__ void reduction_smem(int * input, int * temp, int size)
{
	// add shared memory here
	// allocate shared memory as large as # threads in a thead block
	__shared__ int smem[BLOCK_SIZE];
	int tid = threadIdx.x;
	int * i_data = input + blockDim.x * blockIdx.x;

	// first transfer data from global memory to shared memory
	// note that here only copy data within one thread block range
	// therefore, only part can be replaced in later code is warp unrolling
	// block-size unrolling cannot be used due to no data was transferred
	smem[tid] = i_data[tid];

	// make sure data transfer is complete before any further operation
	// sync all threads within this thread block
	__syncthreads();

	//manual unrolling depending on block size
	// INCORRECT： these parts cannot use shared memory since no data is copied!!
	// actually it is fine, for example, block size=1024, then no exceed
	// or block size=128, then these condition checks won't happen
	// therefore, we should improve performance by replacing i_data with smem
	if (iblock_size >= 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];

	__syncthreads();

	if (iblock_size >= 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];

	__syncthreads();

	if (iblock_size >= 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];

	__syncthreads();

	if (iblock_size >= 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];

	__syncthreads();

	//unrolling warp
	if (tid < 32)
	{
		// only difference is here
		// use shared memory instead of global memory
		volatile int * vsmem = smem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		// this is also different, need to access global memory, directly should use shared memory
		temp[blockIdx.x] = i_data[0];
	}
}


int main(int argc, char ** argv)
{
   printf("Running parallel reduction with complete unrolling kernel \n");

	int kernel_index = 0;

	if (argc > 1)
	{
		kernel_index = 1;
	}

	int size = 1 << 22;
	// int size = 1 << 27;
	int byte_size = size * sizeof(int);
	int block_size = BLOCK_SIZE;

	int * h_input, *h_ref;
	h_input = (int*)malloc(byte_size);

	initialize(h_input, size);

	clock_t cpu_start, cpu_end, gpu_start, gpu_end;

	cpu_start = clock();
	int cpu_result = reduction_cpu(h_input, size);
	cpu_end = clock();

	printf("CPU execution time : %4.6f \n",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));


	// printf("GPU \n");
	// gpu_start = clock();

	dim3 block(block_size);
	dim3 grid((size / block_size));

	int temp_array_byte_size = sizeof(int)* grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int * d_input, *d_temp;
	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	printf("GPU \n");
	gpu_start = clock();

	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
		hipMemcpyHostToDevice));
	if (kernel_index == 0)
	{
		printf("Global memory version - Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

		switch (block_size)
		{
		case 1024:
			reduction_gmem_benchmark <1024> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 512:
			reduction_gmem_benchmark <512> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 256:
			reduction_gmem_benchmark <256> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 128:
			reduction_gmem_benchmark <128> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 64:
			reduction_gmem_benchmark <64> << < grid, block >> > (d_input, d_temp, size);
			break;
		}
	}
	else
	{
		printf("Shared memory version -Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

		switch (block_size)
		{
		case 1024:
			reduction_smem <1024> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 512:
			reduction_smem <512> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 256:
			reduction_smem <256> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 128:
			reduction_smem <128> << < grid, block >> > (d_input, d_temp, size);
			break;
		case 64:
			reduction_smem <64> << < grid, block >> > (d_input, d_temp, size);
			break;
		}
	}

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}

	gpu_end = clock();
	//compare_results(gpu_result, cpu_result);
	print_time_using_host_clock(gpu_start, gpu_end);
	printf("\n");

	gpuErrchk(hipFree(d_input));
	gpuErrchk(hipFree(d_temp));
	free(h_input);
	free(h_ref);

	gpuErrchk(hipDeviceReset());
	return 0;
}